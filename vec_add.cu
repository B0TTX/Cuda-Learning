#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 100000000
#define MAX_ERR 1e-6

__global__ void vecAdd(float *out, float *a, float *b, int n)
{
	for(int i=0; i<n; i++)
	{
		out[i] = a[i] + b[i];
	}
}


int main()
{
	float *a, *b, *out;
	float *x, *y, *z;

	a = (float*)malloc(sizeof(float) * N);
	b = (float*)malloc(sizeof(float) * N);
	out = (float*)malloc(sizeof(float) * N);


	for(int i=0; i<N; i++)
	{
		a[i] = 1.0f;
		b[i] = 2.0f;
	}

	hipMalloc((void**)&x, sizeof(float) * N);
	hipMalloc((void**)&y, sizeof(float) * N);
	hipMalloc((void**)&z, sizeof(float) * N);

	hipMemcpy(x, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(y, b, sizeof(float) * N, hipMemcpyHostToDevice);

	vecAdd<<<1,1>>>(z, x, y, N);

	hipMemcpy(out, z, sizeof(float) * N, hipMemcpyDeviceToHost);

	hipFree(x);
	hipFree(y);
	hipFree(z);

	free(a);
	free(b);
	free(out);
}
